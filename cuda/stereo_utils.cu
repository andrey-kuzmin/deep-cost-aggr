
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_math_constants.h>
#include <stdint.h>
#include <unistd.h>

__global__ void census(float *x0, float *x1, float *output, int size, int num_channels, int size2, int size3, int wnd_half, float bnd_const)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int id = blockId * blockDim.x + threadIdx.x;

	if (id < size) {
		
		int x = blockIdx.x;
		int y = blockIdx.y;
		int d = -threadIdx.x;

		float dist;
		if (0 <= x + d && x + d < size3) {
			dist = 0;
			for (int i = 0; i < num_channels; i++) {
				int ind_p = (i * size2 + y) * size3 + x;
				for (int yy = y - wnd_half; yy <= y + wnd_half; yy++) {
					for (int xx = x - wnd_half; xx <= x + wnd_half; xx++) {
						if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
							int ind_q = (i * size2 + yy) * size3 + xx;
							if ((x0[ind_q] < x0[ind_p]) != (x1[ind_q + d] < x1[ind_p + d])) {
								dist++;
							}
						} else {
							dist++;
						}
					}
				}
			}
			dist /= num_channels;
		} else {
			dist = bnd_const;
		}
		output[id] = dist;
	}
}

__global__ void sad_color(float *x0, float *x1, float *output, int size, int size2, int size3, int wnd_half, float bnd_const)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int id = blockId * blockDim.x + threadIdx.x;
    
    int num_channels = 1;
    
	if (id < size) {
		int x = blockIdx.x;
		int y = blockIdx.y;
		int d = -threadIdx.x;

		float dist;
		if (0 <= x + d && x + d < size3) {
			dist = 0;
			for (int i = 0; i < num_channels; i++) {
				int ind_p = (i * size2 + y) * size3 + x;
				for (int yy = y - wnd_half; yy <= y + wnd_half; yy++) {
					for (int xx = x - wnd_half; xx <= x + wnd_half; xx++) {
	         			if (0 <= xx && xx < size3 && 0 <= xx + d && xx + d < size3 && 0 <= yy && yy < size2) {
			    				int ind_q = (i * size2 + yy) * size3 + xx;
    			    		    dist += abs(x0[ind_p] - x1[ind_q + d]);
    			    	}
					}
				}
			}
		} else {
			dist = bnd_const;
		}
		output[id] = dist;
	}
}

__global__ void linear_comb(float *inp0, float *inp1, float *output, int size, float alpha, float beta)
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int id = blockId * blockDim.x + threadIdx.x;
    
    if (id < size) {
        output[id] = alpha * inp0[id] + beta * inp1[id];
    } 
    
    __syncthreads();
}

__global__ void outlier_detection(float *d0, float *d1, float *outlier, int size, int dim3, int disp_max)
{
	int id = blockIdx.y * gridDim.x + blockIdx.x;
	if (id < size) {
		int x = id % dim3;
		int d0i = d0[id];
		if (x - d0i < 0) {
			//assert(0);
			outlier[id] = 1;
		} else if (abs(d0[id] - d1[id - d0i]) < 1.1) {
			outlier[id] = 0; /* match */
		} else {
			outlier[id] = 1; /* occlusion */
			for (int d = 0; d < disp_max; d++) {
				if (x - d >= 0 && abs(d - d1[id - d]) < 1.1) {
					outlier[id] = 2; /* mismatch */
					break;
				}
			}
		}
	}
}

__device__ void sort(float *x, int n)
{
	for (int i = 0; i < n - 1; i++) {
		int min = i;
		for (int j = i + 1; j < n; j++) {
			if (x[j] < x[min]) {
				min = j;
			}
		}
		float tmp = x[min];
		x[min] = x[i];
		x[i] = tmp;
	}
}

__global__ void interpolate_mismatch(float *d0, float *outlier, float *out, int size, int dim2, int dim3)
{
	const float dir[] = {
		0	,  1,
		-0.5,  1,
		-1	,  1,
		-1	,  0.5,
		-1	,  0,
		-1	, -0.5,
		-1	, -1,
		-0.5, -1,
		0	, -1,
		0.5 , -1,
		1	, -1,
		1	, -0.5,
		1	,  0,
		1	,  0.5,
		1	,  1,
		0.5 ,  1
	};

	int id = blockIdx.y * gridDim.x + blockIdx.x;
	if (id < size) {
		if (outlier[id] != 2) {
			out[id] = d0[id];
			return;
		}

		float vals[16];
		int vals_size = 0;

		int x = id % dim3;
		int y = id / dim3;
		for (int d = 0; d < 16; d++) {
			float dx = dir[2 * d];
			float dy = dir[2 * d + 1];
			float xx = x;
			float yy = y;
			int xx_i = round(xx);
			int yy_i = round(yy);
			while (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3 && outlier[yy_i * dim3 + xx_i] == 2) {
				xx += dx;
				yy += dy;
				xx_i = round(xx);
				yy_i = round(yy);
			}

			int ind = yy_i * dim3 + xx_i;
			if (0 <= yy_i && yy_i < dim2 && 0 <= xx_i && xx_i < dim3) {
				assert(outlier[ind] != 2);
				vals[vals_size++] = d0[ind];
			}
		}
		assert(vals_size > 0);
		sort(vals, vals_size);
		out[id] = vals[vals_size / 2];
	}
}

__global__ void interpolate_occlusion(float *d0, float *outlier, float *out, int size, int dim3)
{
	int id = blockIdx.y * gridDim.x + blockIdx.x;
	if (id < size) {
		if (outlier[id] != 1) {
			out[id] = d0[id];
			return;
		}
		int x = id % dim3;

		int dx = 0;
		while (x + dx >= 0 && outlier[id + dx] != 0) {
		    dx--;
		}
		if (x + dx < 0) {
			dx = 0;
			while (x + dx < dim3 && outlier[id + dx] != 0) {
				dx++;
			}
		}
		if (x + dx < dim3) {
			out[id] = d0[id + dx];
		} else {
			out[id] = d0[id];
		}
	}
}

__global__ void dtransform_lr(
    float* output, float* weight,
    const int height, const int width, const int channels) 
{
  //id_e = (y * width + x) * channels + z;
  //id_w = y * w + x
  
  int ind = 0;
  int ind_prev = 0;
  
  float omega = 0.0;
  
  int i_w = 0;
  for (i_w = 1; i_w < width; i_w++)
  {
      ind = (blockIdx.x * width + i_w) * channels + threadIdx.x;
      ind_prev = (blockIdx.x * width + i_w - 1) * channels + threadIdx.x;
      
      omega = weight[blockIdx.x * width + i_w];
      
      output[ind] = (1.0 - omega) * output[ind] + omega * output[ind_prev];
      
  }
  
  for (i_w = width-2; i_w >= 0; i_w--)
  {
      ind = (blockIdx.x * width + i_w) * channels + threadIdx.x;
      ind_prev = (blockIdx.x * width + i_w + 1) * channels + threadIdx.x;
      
      omega = weight[blockIdx.x * width + i_w];
      
      output[ind] = (1.0 - omega) * output[ind] + omega * output[ind_prev];
  }
}

__global__ void dtransform_ud(
    float* output, float* weight,
    const int height, const int width, const int channels) 
{
  //id_e = (y * width + x) * channels + z;
  //id_w = y * w + x
  
  int ind = 0;
  int ind_prev = 0;
  
  float omega = 0.0;
  
  int i_h = 0;
  for (i_h = 1; i_h < height; i_h++)
  {
      ind = (i_h * width + blockIdx.x) * channels + threadIdx.x;
      ind_prev = ((i_h-1) * width + blockIdx.x) * channels + threadIdx.x;
      
      omega = weight[i_h * width + blockIdx.x];
      
      output[ind] = (1.0 - omega) * output[ind] + omega * output[ind_prev];
      
  }
  
  for (i_h = height-2; i_h >= 0; i_h--)
  {
      ind = (i_h * width + blockIdx.x) * channels + threadIdx.x;
      ind_prev = ((i_h+1) * width + blockIdx.x) * channels + threadIdx.x;
      
      omega = weight[i_h * width + blockIdx.x];
      
      output[ind] = (1.0 - omega) * output[ind] + omega * output[ind_prev];
  }
}





